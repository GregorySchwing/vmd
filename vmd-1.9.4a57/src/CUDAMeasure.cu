/***************************************************************************
 *cr
 *cr            (C) Copyright 1995-2019 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
/***************************************************************************
 * RCS INFORMATION:
 *
 *      $RCSfile: CUDAMeasureRDF.cu,v $
 *      $Author: johns $        $Locker:  $             $State: Exp $
 *      $Revision: 1.25 $      $Date: 2011/01/18 15:32:28 $
 *
 ***************************************************************************/
/**
 * \file CUDAMeasure.cu
 * \brief CUDA accelerated trajectory and structure analysis routines.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "Inform.h"
#include "utilities.h"
#include "WKFThreads.h"
#include "WKFUtils.h"
#include "CUDAKernels.h" 

