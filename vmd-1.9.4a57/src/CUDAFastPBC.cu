#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 1995-2019 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
/***************************************************************************
 * RCS INFORMATION:
 *
 *      $RCSfile: CUDAFastPBC.cu,v $
 *      $Author: johns $        $Locker:  $             $State: Exp $
 *      $Revision: 1.8 $       $Date: 2021/12/21 05:36:37 $
 *
 ***************************************************************************/
/**
 * \file CUDAFastPBC.cu
 * \brief CUDA kernels for fast PBC wrapping.
 */

#include <stdio.h>

// Uses thrust for vector ops, various scan() reductions, etc.
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/device_vector.h>

#include "FastPBC.h"

__global__ void inverseboxsize (float *boxsize, float* invboxsize) {
	int tid = threadIdx.x;
	if (tid < 3) {
		invboxsize[tid] = 1.0 / boxsize[tid];
	}
}

/*
// This is an inefficient kernel. Much slower than the one that replaced 
// it below. (~100 us)
__global__ void repositionfragments(int fnum, float *pos, int *compoundmap, 
                                    int *indexlist, float *boxsize, 
                                    float *invboxsize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int i, j, k;
	for (int l = tid; l < fnum; l+=blockDim.x * gridDim.x) {
		float ccenter[3];
		int lowbound = compoundmap[l];
		int highbound = compoundmap[l+1];
		i = indexlist[lowbound];
		//Use the first element within the compound as the center.
		for (j=0; j < 3; j++) {
			ccenter[j] = pos[i*3+j];
		}
		//move the compound, wrapping it to be within half a box dimension from the center
		for (k = lowbound; k < highbound; k++ ) {
			i = indexlist[k];
			for (j=0; j < 3; j++) {
				pos[i*3+j] = pos[i*3+j] - (rintf((pos[i*3+j] - ccenter[j]) * invboxsize[j]) * boxsize[j]);
			}
		}
	}
}
*/

// Super-efficient kernel. ~8 us execution time
__global__ void repositionfragments(float *pos, int sellen, int *atomtofragmap,
                                    int *compoundmap, int *indexlist, 
                                    float *boxsize, float *invboxsize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < 3*sellen) {
		int idx = tid / 3;
		int dim = tid % 3;
		int cidx = indexlist[compoundmap[atomtofragmap[idx]]];
		//printf("tid: %d, idx %d, dim %d, cidx %d\n", tid, idx, dim, cidx);
		float center = pos[3 * cidx + dim];
		pos[3*indexlist[idx]+dim] = pos[3*indexlist[idx]+dim] - (rintf((pos[3*indexlist[idx]+dim] - center) * invboxsize[dim]) * boxsize[dim]);
	}
}


__global__ void wrapcompound(float *pos, int sellen, float *center, 
                             int *atomtofragmap, int *indexlist, 
                             float *boxsize, float *invboxsize, 
                             float *fragcenters) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < 3*sellen) {
		int idx = tid / 3;
		int dim = tid % 3;
		int frag = atomtofragmap[idx];
		int aidx = indexlist[idx];
		pos[3*aidx+dim] = pos[3*aidx+dim] - (rintf((fragcenters[3*frag+dim] - center[dim]) * invboxsize[dim]) * boxsize[dim]);
	}
}


__global__ void wrapatomic(float *pos, int sellen, float *center, 
                           int *indexlist, float *boxsize, float *invboxsize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < 3*sellen) {
		int idx = tid / 3;
		int dim = tid % 3;
		int aidx = indexlist[idx];
		pos[3*aidx+dim] = pos[3*aidx+dim] - (rintf((pos[3*aidx+dim] - center[dim]) * invboxsize[dim]) * boxsize[dim]);
	}
}


__global__ void unwrapatomic(float *pos, float *prev, float *prevw, int sellen, 
                             int *indexlist, 
                             float *boxsize, float *invboxsize, float *oldboxsize, float *oldinvboxsize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < 3*sellen) {
		int idx = tid / 3;
		int dim = tid % 3;
		int aidx = indexlist[idx];
		float tmp = pos[3*aidx+dim]; // Holds the wrapped position
		float disp = pos[3*aidx+dim] - prevw[3*aidx+dim]; //Displacement
		pos[3*aidx+dim] = prev[3*aidx+dim] + disp - (rintf((disp) * invboxsize[dim]) * boxsize[dim])
									- (rintf((prevw[3*aidx+dim]-prev[3*aidx+dim])*oldinvboxsize[dim]) * (boxsize[dim]-oldboxsize[dim]));
		prevw[3*aidx+dim] = tmp;//Set the prevw for the next frame.
	}
}


__global__ void fragmentperatom(int fnum, int *compoundmap, 
                                int *atomtofragmap) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < fnum && tid != 0) {
		atomtofragmap[compoundmap[tid]] = 1;
	}
}


// XXX this likely duplicates stuff from prototypes in CUDAMeasure
__global__ void measurecenter(float *pos, float *center, int len, 
                              float *weights, int *weightidx, float *wscale) {
	__shared__ float reduce[96]; //96 is not an arbitrary number. Its divisible by 3! This lets us use
	//aligned memory accesses.
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	float mcenter = 0;
	int dim;
	if (tid < 3*len) {
		int idx = tid / 3;
		dim = tid % 3;
		mcenter = pos[3 * weightidx[idx] + dim] * weights[idx] * (*wscale);
	}
	reduce[threadIdx.x] = mcenter;
	__syncthreads();
	if (threadIdx.x < 48) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 48];
	}
	__syncthreads();
	if (threadIdx.x < 24) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 24];
	}
	__syncthreads();
	if (threadIdx.x < 12) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 12];
	}
	__syncthreads();
	if (threadIdx.x < 3) {
		mcenter = reduce[threadIdx.x] + reduce[threadIdx.x + 3] + reduce[threadIdx.x + 6] + reduce[threadIdx.x + 9];
		atomicAdd(&center[dim], mcenter);
	}
}


// Only differs from measurecenter based on how the weights are indexed.
// Here the expectation is that the full mass array has been passed, 
// so we need to find only specific elements of the weight array.
__global__ void measurecenter_fullmass(float *pos, float *center, int len, 
                                       float *weights, int *weightidx, 
                                       float *wscale) {
	__shared__ float reduce[96]; //96 is not an arbitrary number. Its divisible by 3! This lets us use
	//aligned memory accesses.
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	float mcenter = 0;
	int dim;
	if (tid < 3*len) {
		int idx = tid / 3;
		dim = tid % 3;
		int widx = weightidx[idx];
		mcenter = pos[3 * widx + dim] * weights[widx] * (*wscale);
	}
	reduce[threadIdx.x] = mcenter;
	__syncthreads();
	if (threadIdx.x < 48) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 48];
	}
	__syncthreads();
	if (threadIdx.x < 24) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 24];
	}
	__syncthreads();
	if (threadIdx.x < 12) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 12];
	}
	__syncthreads();
	if (threadIdx.x < 3) {
		mcenter = reduce[threadIdx.x] + reduce[threadIdx.x + 3] + reduce[threadIdx.x + 6] + reduce[threadIdx.x + 9];
		atomicAdd(&center[dim], mcenter);
	}
}


// Harrumph. This kernel is inefficient. Less inefficient than the prettier 
// way of doing it, but this has only 1 kernel call, whereas the other one 
// had as many calls as there were fragments.
__global__ void computefragcenters(float *pos, float *centers, int fnum, 
                                   float *weights, float *wscale, 
                                   int *compoundmap, int *indexlist) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int i, j, k, f;
	float ccenter = 0;
	if (tid < 3*fnum) {
		f = tid / 3;
		j = tid % 3;
		int lowbound = compoundmap[f];
		int highbound = compoundmap[f+1];
		//Find the center of the compound.
		for (k = lowbound; k < highbound; k++ ) {
			i = indexlist[k];
			ccenter += pos[i*3+j] * weights[i] * wscale[f];
		}
		centers[3*f+j] = ccenter ;
	}
}


__global__ void fragwscale(float *fragscales, float *massarr, int fragnum, 
                           int *compoundmap, int *indexlist) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int i, k, f;
	float fragmass = 0;
	if (tid < fragnum) {
		f = tid;
		int lowbound = compoundmap[f];
		int highbound = compoundmap[f+1];
		//Weigh the fragment.
		for (k = lowbound; k < highbound; k++ ) {
			i = indexlist[k];
			fragmass += massarr[i];
		}
		fragscales[tid] = 1.0 / fragmass ;
	}
}

void fpbc_exec_unwrap(Molecule* mol, int first, int last, int sellen, int* indexlist) {
	Timestep *ts;
	int f;
	const int threads = 128;
	float *pos;
	float *gpupos;
	float *gpuprevu;
	float *gpuprevw;
	float boxsize[3];
	float *gpuboxsize;
	float *gpuinvboxsize;
	float *gpuoldboxsize;
	float *gpuoldinvboxsize;
	int *gpuindexlist;
	int blocks = (3*sellen + threads - 1) / threads;
	hipHostRegister(indexlist, sizeof(int) * sellen,0);
	hipHostRegister(boxsize, sizeof(float) * 3,0);

	hipMalloc((void**) &gpupos, sizeof(float) * 3*mol->nAtoms);
	hipMalloc((void**) &gpuprevw, sizeof(float) * 3*mol->nAtoms);
	hipMalloc((void**) &gpuprevu, sizeof(float) * 3*mol->nAtoms);
	hipMalloc((void**) &gpuboxsize, sizeof(float) * 3);
	hipMalloc((void**) &gpuinvboxsize, sizeof(float) * 3);
	hipMalloc((void**) &gpuoldboxsize, sizeof(float) * 3);
	hipMalloc((void**) &gpuoldinvboxsize, sizeof(float) * 3);
	hipHostMalloc((void**) &pos, sizeof(float) * 3*mol->nAtoms);
	hipMalloc((void**) &gpuindexlist, sizeof(int) * sellen);
	hipMemcpy(gpuindexlist, indexlist, sizeof(int) * sellen, hipMemcpyHostToDevice);
	f = first;
	ts = mol->get_frame(f);
	//Load in the first frame.
	hipMemcpyAsync(pos, ts->pos, sizeof(float) * 3*mol->nAtoms,hipMemcpyHostToHost);
	hipMemcpyAsync(gpuprevw, pos, sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice);
	hipMemcpyAsync(gpuprevu, gpuprevw, sizeof(float) * 3*mol->nAtoms, hipMemcpyDeviceToDevice);
	boxsize[0] = ts->a_length;
	boxsize[1] = ts->b_length;
	boxsize[2] = ts->c_length;
	hipMemcpyAsync(gpuoldboxsize, boxsize, sizeof(float) * 3, hipMemcpyHostToDevice);
	inverseboxsize<<<1,4>>>(gpuoldboxsize, gpuoldinvboxsize);
	//Do stuff
	for (f=first+1; f<=last; f++) {
		ts = mol->get_frame(f);
		boxsize[0] = ts->a_length;
		boxsize[1] = ts->b_length;
		boxsize[2] = ts->c_length;
		//Block here just so that I don't overwrite our pinned buffer before it is written out to VMD memory.
		hipDeviceSynchronize();
		//Buffer should be clear. Copy VMD timestep to our own buffer
		hipMemcpyAsync(pos, ts->pos, sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToHost);
		//Copy pinned host memory to GPU
		hipMemcpyAsync(gpupos, pos, sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice);
		hipMemcpyAsync(gpuboxsize, boxsize, sizeof(float) * 3, hipMemcpyHostToDevice);
		//Do math here.
		inverseboxsize<<<1,4>>>(gpuboxsize, gpuinvboxsize);
		unwrapatomic<<<blocks,threads>>>(gpupos, gpuprevu, gpuprevw, sellen, gpuindexlist, gpuboxsize, gpuinvboxsize, gpuoldboxsize, gpuoldinvboxsize);
		//Copy out to buffer.
		hipMemcpyAsync(pos, gpupos, sizeof(float) * 3*mol->nAtoms, hipMemcpyDeviceToHost);
		hipMemcpyAsync(gpuprevu, gpupos, sizeof(float) * 3*mol->nAtoms, hipMemcpyDeviceToDevice);
		hipMemcpyAsync(gpuoldboxsize, gpuboxsize, sizeof(float) * 3, hipMemcpyDeviceToDevice);
		hipMemcpyAsync(gpuoldinvboxsize, gpuinvboxsize, sizeof(float) * 3, hipMemcpyDeviceToDevice);
		//Copy back to VMD.
		hipMemcpyAsync(ts->pos, pos, sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToHost);
	}
	//Wait for outstanding memory transfers.
	hipDeviceSynchronize();

	//Cleanup
	hipFree(gpupos);
	hipFree(gpuprevw);
	hipFree(gpuprevu);
	hipFree(gpuboxsize);
	hipFree(gpuinvboxsize);
	hipFree(gpuoldboxsize);
	hipFree(gpuoldinvboxsize);
	hipHostFree(pos);

	hipHostUnregister(indexlist);
	hipHostUnregister(boxsize);
	hipFree(gpuindexlist);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and fallback to CPU
		printf("CUDA error: %s\n", hipGetErrorString(error));
		printf("Reverting to CPU algorithm\n");
		fpbc_exec_unwrap_cpu(mol, first, last, sellen, indexlist);
	}
}


void fpbc_exec_wrapcompound(Molecule* mol, int first, int last, int fnum, int *compoundmap, int sellen, int* indexlist, float* weights, AtomSel* csel, float* center, float* massarr) {
	//Declare variables.
	int f, i, j;
	Timestep *ts;
	const int nStreams = 4;
	const int threads = 128;
	float *pos[nStreams];
	float *gpupos[nStreams];
	float boxsize[3];
	float *gpuboxsize[nStreams];
	float *gpucenters[nStreams];
	float *gpuinvboxsize[nStreams];
	float *gpuweights;
	float *wscale;
	float *gpufragweight;
	int *gpuweightidx;
	int *gpuindexlist;
	int *gpuatomtofragmap;
	int *gpucompoundmap;
	float *gpufragcenters[nStreams];
	int blocks_frag = (fnum + threads - 1) / threads;
	hipStream_t stream[nStreams];
	//Allocate memory for static things (weight sums, maps, etc.)
	hipMalloc((void**) &gpuatomtofragmap, sizeof(int) * sellen);
	hipMalloc((void**) &wscale, sizeof(float));
	hipMemset(gpuatomtofragmap, 0, sizeof(int) * sellen);
	hipMalloc((void**) &gpucompoundmap, sizeof(int) * (fnum+1));
	hipMalloc((void**) &gpuindexlist, sizeof(int) * sellen);
	hipMalloc((void**) &gpuweights, sizeof(float) * mol->nAtoms);
	hipMalloc((void**) &gpufragweight, sizeof(float) * fnum);
	hipHostRegister(compoundmap, sizeof(int) * (fnum+1),0);
	hipHostRegister(indexlist, sizeof(int) * sellen,0);
	hipMemcpy(gpuweights, massarr, sizeof(float) * mol->nAtoms, hipMemcpyHostToDevice);//Unlike in wrapatomic, we'll pass over the full mass array to the GPU, since odds are we'll need it.
	hipMemcpy(gpucompoundmap, compoundmap, sizeof(int) * (fnum+1), hipMemcpyHostToDevice);
	hipMemcpy(gpuindexlist, indexlist, sizeof(int) * sellen, hipMemcpyHostToDevice);
	//Make the atomtofragmap by setting elements to 1 and then doing a scan.
	fragmentperatom<<<blocks_frag, threads>>>(fnum, gpucompoundmap, gpuatomtofragmap);//Setup the gpu per atom map.
	thrust::inclusive_scan(thrust::device_ptr<int>(gpuatomtofragmap), thrust::device_ptr<int>(gpuatomtofragmap + sellen), thrust::device_ptr<int>(gpuatomtofragmap));
	//Get the mass per fragment (for scaling/finding center of mass for everything.)
	fragwscale<<<blocks_frag, threads>>>(gpufragweight, gpuweights, fnum, gpucompoundmap, gpuindexlist);
	hipHostRegister(boxsize, sizeof(float)*3,0);
	if (csel != NULL) {
		hipMalloc((void**) &gpuweightidx, sizeof(int) * csel->selected);
		int *weightidx = new int[csel->selected];
		j=0;
		for (i=csel->firstsel; i<=csel->lastsel; i++) {
			if (csel->on[i]) {
				weightidx[j++] = i;
			}
		}
		hipMemcpy(gpuweightidx, weightidx, sizeof(int) * csel->selected, hipMemcpyHostToDevice);
		thrust::device_vector<int> ids (thrust::device_ptr<int>(gpuweightidx), thrust::device_ptr<int>(gpuweightidx+csel->selected));
		thrust::device_vector<float> mass (thrust::device_ptr<float>(gpuweights), thrust::device_ptr<float>(gpuweights+mol->nAtoms));
		float tmp = 1.0f / thrust::reduce(thrust::make_permutation_iterator(mass.begin(), ids.begin()),
			thrust::make_permutation_iterator(mass.end(), ids.end()), 0, thrust::plus<float>());
		hipMemcpy(wscale, &tmp, sizeof(float), hipMemcpyHostToDevice);
		delete [] weightidx;
	}
	//Allocate memory and create streams for per-frame changables.
	for (f = 0; f< nStreams; f++) {
		hipStreamCreate(&stream[f]);
		hipMalloc((void**) &gpupos[f], sizeof(float) * 3*mol->nAtoms);
		hipMalloc((void**) &gpuboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpuinvboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpucenters[f], sizeof(float) * 3);
		hipMemcpyAsync(gpucenters[f], center, sizeof(float)*3, hipMemcpyHostToDevice, stream[f]);
		hipMalloc((void**) &gpufragcenters[f], sizeof(float) * 3 * fnum);
		hipHostMalloc(&pos[f], sizeof(float) * 3*mol->nAtoms);
	}
	hipDeviceSynchronize();
	//Start looping over the frames.
	int blocks = (3*sellen + threads - 1) / threads;
	blocks_frag = (3*fnum + threads - 1) / threads;
	for (f=first; f<=last; f++) {
		ts = mol->get_frame(f);
		boxsize[0] = ts->a_length;
		boxsize[1] = ts->b_length;
		boxsize[2] = ts->c_length;
		//Block here just so that I don't overwrite a buffer.
		hipStreamSynchronize(stream[f%nStreams]);
		//Buffer should be clear. Copy VMD timestep to our own buffer
		hipMemcpyAsync(pos[f%nStreams], ts->pos, sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToHost, stream[f%nStreams]);
		hipMemcpyAsync(gpupos[f%nStreams], pos[f%nStreams], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice, stream[f%nStreams]);
		hipMemcpyAsync(gpuboxsize[f%nStreams], boxsize, sizeof(float)*3, hipMemcpyHostToDevice, stream[f%nStreams]);
		//Do math here.
		inverseboxsize<<<1,4,0,stream[f%nStreams]>>>(gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		if (csel != NULL) {
			hipMemsetAsync(gpucenters[f%nStreams],0, 3 * sizeof(float), stream[f%nStreams]);
			//Measure the center of the selection if one is provided. Put it into the 3-vector gpucenters.
			//To exploit some of the symmetry of the problem, pick a blocksize that is a multiple of 3, and preferably
			//also a multiple of the warpsize (96 is good!)
			measurecenter_fullmass<<<(3*csel->selected + 95) / 96, 96, 0, stream[f%nStreams]>>>(gpupos[f%nStreams], gpucenters[f%nStreams], csel->selected, gpuweights, gpuweightidx, wscale);
		}
		//Fragment centers need to be determined.
		//TODO: make this not suck. At the moment, I think this is the biggest bottleneck.
		computefragcenters<<<blocks_frag,threads,0,stream[f%nStreams]>>>(gpupos[f%nStreams], gpufragcenters[f%nStreams], fnum, gpuweights, gpufragweight, gpucompoundmap, gpuindexlist);
		//Wrap.
		wrapcompound<<<blocks, threads, 0, stream[f%nStreams]>>> (gpupos[f%nStreams], sellen, gpucenters[f%nStreams], gpuatomtofragmap, gpuindexlist, gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams], gpufragcenters[f%nStreams]);
		//Copy back.
		hipMemcpyAsync(pos[f%nStreams], gpupos[f%nStreams], sizeof(float) * 3 *mol->nAtoms, hipMemcpyDeviceToHost, stream[f%nStreams]);
		//Copy back to VMD.
		hipMemcpyAsync(ts->pos, pos[f%nStreams], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToHost, stream[f%nStreams]);
	}
	//Cleanup
	//Wait for outstanding memory transfers.
	hipDeviceSynchronize();
	//Free memory.
	hipHostUnregister(boxsize);
	hipHostUnregister(compoundmap);
	hipHostUnregister(indexlist);
	hipFree(gpucompoundmap);
	hipFree(gpuindexlist);
	hipFree(gpuatomtofragmap);
	for (f = 0; f< nStreams; f++) {
		hipStreamDestroy(stream[f]);
		hipFree(gpupos[f]);
		hipFree(gpuboxsize[f]);
		hipFree(gpuinvboxsize[f]);
		hipHostFree(pos[f]);
	}
	
	hipFree(wscale);
	hipFree(gpufragweight);
	hipFree(gpuweights);
	if (csel != NULL) {
		hipFree(gpuweightidx);
	}
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and fallback to CPU
		printf("CUDA error: %s\n", hipGetErrorString(error));
		printf("Reverting to CPU algorithm\n");
		fpbc_exec_wrapcompound_cpu(mol, first, last, fnum, compoundmap, sellen, indexlist, weights, csel, center, massarr);
	}
}


void fpbc_exec_wrapatomic(Molecule* mol, int first, int last, int sellen, int* indexlist, 
	float* weights, AtomSel* csel, float* center) {
	int f, i, j;
	Timestep *ts;
	const int nStreams = 4;
	const int threads = 128;
	float *gpupos[nStreams];
	float boxsize[3];
	float *pos[nStreams];
	float *gpuboxsize[nStreams];
	float *gpucenters[nStreams];
	float *gpuinvboxsize[nStreams];
	float *gpuweights;
	float *wscale;
	hipMalloc((void**) &wscale, sizeof(float));
	int *gpuweightidx;
	int *gpuindexlist;
	//Prepare GPU memory and streams
	hipStream_t stream[nStreams];
	hipMalloc((void**) &gpuindexlist, sizeof(int) * sellen);
	hipMemcpy(gpuindexlist, indexlist, sizeof(int) * sellen, hipMemcpyHostToDevice);
	hipHostRegister(center, sizeof(float) * 3,0);
	hipHostRegister(boxsize, sizeof(float) * 3,0);
	for (f = 0; f< nStreams; f++) {
		hipStreamCreate(&stream[f]);
		hipMalloc((void**) &gpupos[f], sizeof(float) * 3*mol->nAtoms);
		hipMalloc((void**) &gpuboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpuinvboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpucenters[f], sizeof(float) * 3);
		hipMemcpyAsync(gpucenters[f], center, sizeof(float)*3, hipMemcpyHostToDevice, stream[f]);
		hipHostMalloc(&pos[f], sizeof(float) * 3*mol->nAtoms);
	}
	if (csel != NULL) {
		hipMalloc((void**) &gpuweights, sizeof(float) * csel->selected);
		hipMalloc((void**) &gpuweightidx, sizeof(int) * csel->selected);
		hipMemcpy(gpuweights, weights, sizeof(float) * csel->selected, hipMemcpyHostToDevice);
		int *weightidx = new int[csel->selected];
		j=0;
		for (i=csel->firstsel; i<=csel->lastsel; i++) {
			if (csel->on[i]) {
				weightidx[j++] = i;
			}
		}
		hipMemcpy(gpuweightidx, weightidx, sizeof(int) * csel->selected, hipMemcpyHostToDevice);
		float tmp = 1.0f / thrust::reduce(thrust::device_ptr<float>(gpuweights), thrust::device_ptr<float>(gpuweights + csel->selected), 0, thrust::plus<float>());
		hipMemcpy(wscale, &tmp, sizeof(float), hipMemcpyHostToDevice);
		delete [] weightidx;
	}
	int blocks = (3*sellen + threads - 1) / threads;
	for (f=first; f<=last; f++) {
		ts = mol->get_frame(f);
		boxsize[0] = ts->a_length;
		boxsize[1] = ts->b_length;
		boxsize[2] = ts->c_length;
		//Block here just so that I don't overwrite a buffer.
		hipStreamSynchronize(stream[f%nStreams]);
		hipMemcpyAsync(pos[f%nStreams], ts->pos, sizeof(float) * 3*mol->nAtoms,hipMemcpyHostToHost, stream[f%nStreams]);
		hipMemcpyAsync(gpupos[f%nStreams], pos[f%nStreams], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice, stream[f%nStreams]);
		hipMemcpyAsync(gpuboxsize[f%nStreams], boxsize, sizeof(float)*3, hipMemcpyHostToDevice, stream[f%nStreams]);
		//Do math here.
		inverseboxsize<<<1,4,0,stream[f%nStreams]>>>(gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		if (csel != NULL) {
			hipMemsetAsync(gpucenters[f%nStreams],0, 3 * sizeof(float), stream[f%nStreams]);
			//Measure the center of the selection if one is provided. Put it into the 3-vector gpucenters.
			//To exploit some of the symmetry of the problem, pick a blocksize that is a multiple of 3, and preferably
			//also a multiple of the warpsize (96 is good!)
			measurecenter<<<(3*csel->selected + 95) / 96, 96, 0, stream[f%nStreams]>>>(gpupos[f%nStreams], gpucenters[f%nStreams], csel->selected, gpuweights, gpuweightidx, wscale);
		}
		//Wrap.
		wrapatomic<<<blocks, threads, 0, stream[f%nStreams]>>> (gpupos[f%nStreams], sellen, gpucenters[f%nStreams], gpuindexlist, gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		//Copy back.
		hipMemcpyAsync(pos[f%nStreams], gpupos[f%nStreams], sizeof(float) * 3 *mol->nAtoms, hipMemcpyDeviceToHost, stream[f%nStreams]);
		hipMemcpyAsync(ts->pos, pos[f%nStreams], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToHost, stream[f%nStreams]);
	}
	//Wait for outstanding memory transfers.
	hipDeviceSynchronize();

	hipHostUnregister(boxsize);
	hipHostUnregister(center);
	hipFree(gpuindexlist);
	for (f = 0; f< nStreams; f++) {
		hipStreamDestroy(stream[f]);
		hipFree(gpupos[f]);
		hipFree(gpuboxsize[f]);
		hipFree(gpuinvboxsize[f]);
		hipFree(gpucenters[f]);
		hipHostFree(pos[f]);
	}
	
	if (csel != NULL) {
		hipFree(gpuweights);
		hipFree(gpuweightidx);
	}
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and fallback to CPU
		printf("CUDA error: %s\n", hipGetErrorString(error));
		printf("Reverting to CPU algorithm\n");
		fpbc_exec_wrapatomic_cpu(mol, first, last, sellen, indexlist, weights, csel, center);
	}
}


void fpbc_exec_join(Molecule* mol, int first, int last, int fnum, int *compoundmap, int sellen, int* indexlist) {
	int f;
	
	Timestep *ts;
	const int nStreams = 4;
	const int threads = 128;
	float *gpupos[nStreams];
	float *pos[nStreams];
	float boxsize[3];
	float *gpuboxsize[nStreams];
	float *gpuinvboxsize[nStreams];
	int *gpucompoundmap;
	int *gpuatomtofragmap;
	int *gpuindexlist;
	int blocks = (fnum + threads - 1) / threads;
	hipStream_t stream[nStreams];
	hipMalloc((void**) &gpuatomtofragmap, sizeof(int) * sellen);
	hipMemset(gpuatomtofragmap, 0, sizeof(int) * sellen);
	hipMalloc((void**) &gpucompoundmap, sizeof(int) * (fnum+1));
	hipMalloc((void**) &gpuindexlist, sizeof(int) * sellen);
	hipHostRegister(compoundmap, sizeof(int) * (fnum+1),0);
	hipHostRegister(indexlist, sizeof(int) * sellen,0);
	hipMemcpy(gpucompoundmap, compoundmap, sizeof(int) * (fnum+1), hipMemcpyHostToDevice);
	hipMemcpy(gpuindexlist, indexlist, sizeof(int) * sellen, hipMemcpyHostToDevice);
	fragmentperatom<<<blocks, threads>>>(fnum, gpucompoundmap, gpuatomtofragmap);//Setup the gpu per atom map.
	thrust::inclusive_scan(thrust::device_ptr<int>(gpuatomtofragmap), thrust::device_ptr<int>(gpuatomtofragmap + sellen), thrust::device_ptr<int>(gpuatomtofragmap));
	for (f = 0; f< nStreams; f++) {
		hipStreamCreate(&stream[f]);
		hipMalloc((void**) &gpupos[f], sizeof(float) * 3*mol->nAtoms);
		hipMalloc((void**) &gpuboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpuinvboxsize[f], sizeof(float) * 3);
		hipHostMalloc(&pos[f], sizeof(float) * 3*mol->nAtoms);
	}
	hipHostRegister(boxsize, sizeof(float)*3,0);
	
	//Make sure the gpuatomtofragmap is set before proceeding.
	hipDeviceSynchronize();
	blocks = (3*sellen + threads - 1) / threads;
	for (f = first; f <= last; f++) {
		ts = mol->get_frame(f);
		boxsize[0] = ts->a_length;
		boxsize[1] = ts->b_length;
		boxsize[2] = ts->c_length;
		//Block here just so that I don't overwrite a buffer.
		hipStreamSynchronize(stream[f%nStreams]);
		hipMemcpyAsync(pos[f%nStreams], ts->pos, sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToHost, stream[f%nStreams]);
		//Copy pinned host memory to GPU
		hipMemcpyAsync(gpupos[f%nStreams], pos[f%nStreams], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice, stream[f%nStreams]);
		hipMemcpyAsync(gpuboxsize[f%nStreams], boxsize, sizeof(float)*3, hipMemcpyHostToDevice, stream[f%nStreams]);
		//Do math here.
		inverseboxsize<<<1,4,0,stream[f%nStreams]>>>(gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		repositionfragments<<<blocks,threads, 0, stream[f%nStreams]>>>(gpupos[f%nStreams], sellen, gpuatomtofragmap,
			gpucompoundmap, gpuindexlist, gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		//Copy back.
		hipMemcpyAsync(pos[f%nStreams], gpupos[f%nStreams], sizeof(float) * 3*mol->nAtoms, hipMemcpyDeviceToHost,stream[f%nStreams]);
		//Copy back to VMD.
		hipMemcpyAsync(ts->pos, pos[f%nStreams], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToHost, stream[f%nStreams]);
	}
	//Wait for outstanding memory transfers.
	hipDeviceSynchronize();

	hipHostUnregister(boxsize);
	hipHostUnregister(indexlist);
	hipHostUnregister(compoundmap);
	hipFree(gpucompoundmap);
	hipFree(gpuindexlist);
	hipFree(gpuatomtofragmap);
	for (f = 0; f< nStreams; f++) {
		hipStreamDestroy(stream[f]);
		hipFree(gpupos[f]);
		hipFree(gpuboxsize[f]);
		hipFree(gpuinvboxsize[f]);
		hipHostFree(pos[f]);
	}
	
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and fallback to CPU
		printf("CUDA error: %s\n", hipGetErrorString(error));
		printf("Reverting to CPU algorithm\n");
		fpbc_exec_join(mol, first, last, fnum, compoundmap, sellen, indexlist);
	}
}


void fpbc_exec_recenter(Molecule* mol, int first, int last, int csellen, int* cindexlist, int fnum, int *compoundmap, int sellen, int* indexlist, float* weights, AtomSel* csel, float* massarr) {
	//The basic idea here is to pass the data back and forth only once while both unwrapping and rewrapping the trajectory.
	Timestep *ts;
	int f;
	const int threads = 128;
	float *pos;
	float *gpupos;
	float *gpuprevu;
	float *gpuprevw;
	float boxsize[3];
	float *gpuboxsize;
	float *gpucenters;
	float *gpuinvboxsize;
	float *gpuoldboxsize;
	float *gpuoldinvboxsize;
	float *gpufragcenters;
	float *wscale;
	hipMalloc((void**) &wscale, sizeof(float));
	float *gpuweights;
	int *gpuweightidx;
	int *gpuindexlist;
	float *gpufragweight;
	int *gpuatomtofragmap;
	int *gpucompoundmap;
	hipHostRegister(indexlist, sizeof(int) * sellen,0);
	hipHostRegister(cindexlist, sizeof(int) * csellen,0);
	hipHostRegister(boxsize, sizeof(float) * 3,0);
	int blocks = (3*sellen + threads - 1) / threads;
	int blocks_frag = (fnum + threads - 1) / threads;
	hipMalloc((void**) &gpupos, sizeof(float) * 3*mol->nAtoms);
	hipMalloc((void**) &gpuprevw, sizeof(float) * 3*mol->nAtoms);
	hipMalloc((void**) &gpuprevu, sizeof(float) * 3*mol->nAtoms);
	hipMalloc((void**) &gpuboxsize, sizeof(float) * 3);
	hipMalloc((void**) &gpuinvboxsize, sizeof(float) * 3);
	hipMalloc((void**) &gpuoldboxsize, sizeof(float) * 3);
	hipMalloc((void**) &gpuoldinvboxsize, sizeof(float) * 3);
	hipMalloc((void**) &gpucenters, sizeof(float) * 3);
	if (fnum) {
		hipMalloc((void**) &gpufragcenters, sizeof(float) * 3 * fnum);
	}
	hipHostMalloc((void**) &pos, sizeof(float) * 3*mol->nAtoms);
	float tmp;
	hipMalloc((void**) &gpuindexlist, sizeof(int) * sellen);
	
	hipMemcpy(gpuindexlist, indexlist, sizeof(int) * sellen, hipMemcpyHostToDevice);
	//Deal with computing the weighted center of mass.
	hipMalloc((void**) &gpuweightidx, sizeof(int) * csellen);
	hipMemcpy(gpuweightidx, cindexlist, sizeof(int) * csellen, hipMemcpyHostToDevice);
	if (fnum) {//Compound runs only.
		hipMalloc((void**) &gpuweights, sizeof(float) * mol->nAtoms);
		hipMalloc((void**) &gpufragweight, sizeof(float) * fnum);
		hipHostRegister(compoundmap, sizeof(int) * (fnum+1),0);
		hipMemcpy(gpuweights, massarr, sizeof(float) * mol->nAtoms, hipMemcpyHostToDevice);
		hipMalloc((void**) &gpucompoundmap, sizeof(int) * (fnum+1));
		hipMemcpy(gpucompoundmap, compoundmap, sizeof(int) * (fnum+1), hipMemcpyHostToDevice);
		hipMalloc((void**) &gpuatomtofragmap, sizeof(int) * sellen);
		hipMemset(gpuatomtofragmap, 0, sizeof(int) * sellen);
		fragmentperatom<<<blocks_frag, threads>>>(fnum, gpucompoundmap, gpuatomtofragmap);//Setup the gpu per atom map.
		thrust::inclusive_scan(thrust::device_ptr<int>(gpuatomtofragmap), thrust::device_ptr<int>(gpuatomtofragmap + sellen), thrust::device_ptr<int>(gpuatomtofragmap));
		//Get the mass per fragment (for scaling/finding center of mass for everything.)
		fragwscale<<<blocks_frag, threads>>>(gpufragweight, gpuweights, fnum, gpucompoundmap, gpuindexlist);
		thrust::device_vector<int> ids (thrust::device_ptr<int>(gpuweightidx), thrust::device_ptr<int>(gpuweightidx+csel->selected));
		thrust::device_vector<float> mass (thrust::device_ptr<float>(gpuweights), thrust::device_ptr<float>(gpuweights+mol->nAtoms));
		tmp = 1.0f / thrust::reduce(thrust::make_permutation_iterator(mass.begin(), ids.begin()),
			thrust::make_permutation_iterator(mass.end(), ids.end()), 0, thrust::plus<float>());
		hipDeviceSynchronize();
	}
	else {
		hipMalloc((void**) &gpuweights, sizeof(float) * csel->selected);
		hipMemcpy(gpuweights, weights, sizeof(float) * csel->selected, hipMemcpyHostToDevice);
		tmp = 1.0f / thrust::reduce(thrust::device_ptr<float>(gpuweights), thrust::device_ptr<float>(gpuweights + csel->selected), 0, thrust::plus<float>());
	}
	hipMemcpy(wscale, &tmp, sizeof(float), hipMemcpyHostToDevice);

	//Do stuff
	blocks_frag = (3*fnum + threads - 1) / threads;
	for (f=first; f<=last; f++) {
		ts = mol->get_frame(f);
		if ( f > first) {
			hipMemcpyAsync(gpuoldboxsize, gpuboxsize, sizeof(float) * 3, hipMemcpyDeviceToDevice);
			hipMemcpyAsync(gpuoldinvboxsize, gpuinvboxsize, sizeof(float) * 3, hipMemcpyDeviceToDevice);
		} else {
			hipMemcpyAsync(gpuprevw, pos, sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice);
			hipMemcpyAsync(gpuprevu, gpuprevw, sizeof(float) * 3*mol->nAtoms, hipMemcpyDeviceToDevice);
		}
		
		boxsize[0] = ts->a_length;
		boxsize[1] = ts->b_length;
		boxsize[2] = ts->c_length;
		hipMemcpyAsync(pos, ts->pos, sizeof(float) * 3*mol->nAtoms,hipMemcpyHostToHost);
		hipMemcpyAsync(gpupos, pos, sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice);
		hipMemcpyAsync(gpuboxsize, boxsize, sizeof(float) * 3, hipMemcpyHostToDevice);
		//Do math here.
		inverseboxsize<<<1,4>>>(gpuboxsize, gpuinvboxsize);
		if (f > first) {//These are the ones that also need to be unwrapped.
			//We must wait until the previous stream is done moving atoms around or loading. This part is inherently serial.
			unwrapatomic<<<blocks,threads>>>(gpupos, gpuprevu, gpuprevw, csellen, gpuweightidx, gpuboxsize, gpuinvboxsize, gpuoldboxsize, gpuoldinvboxsize);
			hipMemcpyAsync(gpuprevu, gpupos, sizeof(float) * 3*mol->nAtoms, hipMemcpyDeviceToDevice);
		}
		hipMemsetAsync(gpucenters,0, 3 * sizeof(float));		
		//Compounding will have a non-zero fnum.
		if (fnum) {
			measurecenter_fullmass<<<(3*csel->selected + 95) / 96, 96>>>(gpupos, gpucenters, csel->selected, gpuweights, gpuweightidx, wscale);
			//Wrap.
			wrapcompound<<<blocks, threads>>> (gpupos, sellen, gpucenters, gpuatomtofragmap, gpuindexlist, gpuboxsize, gpuinvboxsize, gpufragcenters);
		}
		else {
			measurecenter<<<(3*csel->selected + 95) / 96, 96>>>(gpupos, gpucenters, csel->selected, gpuweights, gpuweightidx, wscale);
			//Wrap.
			wrapatomic<<<blocks, threads>>> (gpupos, sellen, gpucenters, gpuindexlist, gpuboxsize, gpuinvboxsize);
		}
		//Copy out to buffer.
		hipMemcpyAsync(pos, gpupos, sizeof(float) * 3*mol->nAtoms, hipMemcpyDeviceToHost);
		//Copy back to VMD.
		hipMemcpyAsync(ts->pos, pos, sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToHost);
	}
	//Wait for outstanding memory transfers.
	hipDeviceSynchronize();
	//Cleanup
	hipFree(gpupos);
	hipFree(gpuboxsize);
	hipFree(gpuinvboxsize);
	hipFree(gpucenters);
	if (fnum) {
		hipFree(gpufragcenters);
	}
	hipHostFree(pos);
	hipFree(gpuprevw);
	hipFree(gpuprevu);
	hipFree(gpuoldboxsize);
	hipFree(gpuoldinvboxsize);

	hipHostUnregister(indexlist);
	hipHostUnregister(cindexlist);
	hipHostUnregister(boxsize);
	hipFree(gpuindexlist);
	hipFree(gpuweightidx);
	hipFree(gpuweights);
	hipFree(wscale);
	if (fnum) {
		hipFree(gpucompoundmap);
		hipFree(gpuatomtofragmap);
		hipFree(gpufragweight);
		hipHostUnregister(compoundmap);
	}
	
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and fallback to CPU
		printf("CUDA error: %s\n", hipGetErrorString(error));
		printf("Reverting to CPU algorithm\n");
		fpbc_exec_recenter_cpu(mol, first, last, csellen, cindexlist, fnum, compoundmap, sellen, indexlist, weights, csel, massarr);
	}
}
